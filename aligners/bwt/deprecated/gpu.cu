#include "hip/hip_runtime.h"
__global__ void BWExactIterativeSearchGPU(REF_TYPE *W, SA_TYPE *nW, SA_TYPE *nWe, int *k, int *l, int k_ini, int l_ini, int *C,  int *O,  int sizO) {

	intmax_t i, b; //, pos;
	SA_TYPE k2, l2;
	REF_TYPE val1, val2, val3, val4;
	int siz1, siz2, siz3, siz4;

	int offset  = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int Cshared[4];

	if (threadIdx.x<4) { // Minimum data is a 32 block
		Cshared[threadIdx.x] = C[threadIdx.x];
	}

	__syncthreads();

	k2 = k_ini;
	l2 = l_ini;

	// First block of 4 bases should not be fully filled

	int real_size = nWe[offset];
	int resto = real_size % 4;

	b = W[offset*MAXLINECOMP + nW[offset] - 1];

	switch(resto) {

		case 0:
			val4 = ( b >> 6 ) & 3;
			siz4 = val4*sizO;

			k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
			l2 = Cshared[val4] + O[siz4 + l2 + 1];

		case 3:
			val3 = ( b >> 4 ) & 3;
			siz3 = val3*sizO;

			k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
			l2 = Cshared[val3] + O[siz3 + l2 + 1];

		case 2:
			val2 = ( b >> 2 ) & 3;
			siz2 = val2*sizO;

			k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
			l2 = Cshared[val2] + O[siz2 + l2 + 1];

		case 1:
			val1 = ( b      ) & 3;
			siz1 = val1*sizO;

			k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
			l2 = Cshared[val1] + O[siz1 + l2 + 1];

	}

	__syncthreads();

	for (i=nW[offset]-2; (k2<=l2) && (i>=0); i--) {

		b = W[offset*MAXLINECOMP + i];

		val4 = ( b >> 6 ) & 3;
		siz4 = val4*sizO;

		k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
		l2 = Cshared[val4] + O[siz4 + l2 + 1];

		val3 = ( b >> 4 ) & 3;
		siz3 = val3*sizO;

		k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
		l2 = Cshared[val3] + O[siz3 + l2 + 1];

		val2 = ( b >> 2 ) & 3;
		siz2 = val2*sizO;

		k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
		l2 = Cshared[val2] + O[siz2 + l2 + 1];

		val1 = ( b      ) & 3;
		siz1 = val1*sizO;

		k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
		l2 = Cshared[val1] + O[siz1 + l2 + 1];

	}

	__syncthreads();

	k[offset] = k2;
	l[offset] = l2;

}

__global__ void BWExactIterativeSearchGPURev(REF_TYPE *W, SA_TYPE *nW, SA_TYPE *nWe, SA_TYPE *k, SA_TYPE *l, SA_TYPE k_ini, SA_TYPE l_ini, int *C,  int *O,  int sizO) {

	intmax_t i, b;//, pos;
	SA_TYPE k2, l2;
	char val1, val2, val3, val4;
	int siz1, siz2, siz3, siz4;

	int offset  = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int Cshared[4];

	if (threadIdx.x<4) { // Minimum data is a 32 block
		Cshared[threadIdx.x] = C[threadIdx.x];
	}

	__syncthreads();

	k2 = k_ini;
	l2 = l_ini;

	for (i=0; (k2<=l2) && (i<nW[offset]-1); i++) {

		b = W[offset*MAXLINECOMP + i];

		val1 = ( b      ) & 3;
		siz1 = val1*sizO;

		k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
		l2 = Cshared[val1] + O[siz1 + l2 + 1];

		val2 = ( b >> 2 ) & 3;
		siz2 = val2*sizO;

		k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
		l2 = Cshared[val2] + O[siz2 + l2 + 1];

		val3 = ( b >> 4 ) & 3;
		siz3 = val3*sizO;

		k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
		l2 = Cshared[val3] + O[siz3 + l2 + 1];

		val4 = ( b >> 6 ) & 3;
		siz4 = val4*sizO;

		k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
		l2 = Cshared[val4] + O[siz4 + l2 + 1];

	}

	__syncthreads();

	if (k2<=l2) {

		// Last block of 4 bases should not be fully filled

		int real_size = nWe[offset];
		int resto = real_size % 4;

		b = W[offset*MAXLINECOMP + nW[offset] - 1];

		switch(resto) {

			case 0:

				val1 = ( b      ) & 3;
				siz1 = val1*sizO;

				k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
				l2 = Cshared[val1] + O[siz1 + l2 + 1];

				val2 = ( b >> 2 ) & 3;
				siz2 = val2*sizO;

				k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
				l2 = Cshared[val2] + O[siz2 + l2 + 1];

				val3 = ( b >> 4 ) & 3;
				siz3 = val3*sizO;

				k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
				l2 = Cshared[val3] + O[siz3 + l2 + 1];

				val4 = ( b >> 6 ) & 3;
				siz4 = val4*sizO;

				k2 = Cshared[val4] + O[siz4 + k2    ] + 1;
				l2 = Cshared[val4] + O[siz4 + l2 + 1];

				break;

			case 3:

				val1 = ( b      ) & 3;
				siz1 = val1*sizO;

				k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
				l2 = Cshared[val1] + O[siz1 + l2 + 1];

				val2 = ( b >> 2 ) & 3;
				siz2 = val2*sizO;

				k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
				l2 = Cshared[val2] + O[siz2 + l2 + 1];

				val3 = ( b >> 4 ) & 3;
				siz3 = val3*sizO;

				k2 = Cshared[val3] + O[siz3 + k2    ] + 1;
				l2 = Cshared[val3] + O[siz3 + l2 + 1];

				break;

			case 2:

				val1 = ( b      ) & 3;
				siz1 = val1*sizO;

				k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
				l2 = Cshared[val1] + O[siz1 + l2 + 1];

				val2 = ( b >> 2 ) & 3;
				siz2 = val2*sizO;

				k2 = Cshared[val2] + O[siz2 + k2    ] + 1;
				l2 = Cshared[val2] + O[siz2 + l2 + 1];

				break;

			case 1:
				val1 = ( b      ) & 3;
				siz1 = val1*sizO;

				k2 = Cshared[val1] + O[siz1 + k2    ] + 1;
				l2 = Cshared[val1] + O[siz1 + l2 + 1];

				break;

		}

	}

	__syncthreads();

	k[offset] = k2;
	l[offset] = l2;

}


